#include <iomanip>
#include <iostream>
#include <math.h>
#include <memory.h>
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"


// double 性能会差一些，float 的精度已经足够
// Check error codes for CUDA functions
#define CHECK(call)                                                            \
  {                                                                            \
    hipError_t error = call;                                                  \
    if (error != hipSuccess) {                                                \
      fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                   \
      fprintf(stderr, "code: %d, reason: %s\n", error,                         \
              hipGetErrorString(error));                                      \
    }                                                                          \
  }

using namespace std;

#define IT_NUM 100 // !=1
#define c 2.236068
#define PI 3.1415926535897932
#define Lx 4 // length of X
#define Ly 2

// Block dimensions 调整影响性能
#define BDIMX 32
#define BDIMY 32

double **calloc2D(int x, int y);
void free2D(double **u, int x, int y);

double precise_solution(double x, double y, double total_time) {
  double res = 0.0;
  double coeff = 1.0f;
  // shoule be double loop
  for (int m = 1; m < 100; m += 2) {
    for (int n = 1; n < 100; n += 2) {
      coeff = 1.0 / (m * m * m * n * n * n);
      res += coeff *
             cos(total_time * sqrt(5) * PI / 4 * sqrt(m * m + 4 * n * n)) *
             sin(m * PI * x / 4) * sin(n * PI * y / 2);
    }
  }
  return 0.426050 * res;
}

void print_height(double *h, int Nx, int Ny) {
  for (int j = 0; j < (Ny+1); j++) {
    for (int i = 0; i < (Nx+1); i++) {
      cout << setiosflags(ios::left) << setw(8) << setprecision(10)
           << h[i * (Ny+1) + j] << " ";
    }
    cout << endl;
  }
  cout << endl;
}

__global__ void kernel_2dfd(double *h0, double *h1, double *h2, double dt,
                            double h, int Nx, int Ny) {

  unsigned int it = blockIdx.x * blockDim.x + threadIdx.x;

  if (it >= 1 && it <= IT_NUM - 1) {
    for (int i = 1; i <= Nx - 1; i++) {
      for (int j = 1; j <= Ny - 1; j++) {
        h2[i * Ny + j] =
            2 * h1[i * Ny + j] - h0[i * Ny + j] +
            (c * c * dt * dt / (h * h)) *
                (h1[(i + 1) * Ny + j] + h1[(i - 1) * Ny + j] +
                 h1[i * Ny + j + 1] + h1[i * Ny + j - 1] - 4 * h1[i * Ny + j]);
      }
    }
    // renew h1(t-1) with h1(t) and h1(t) with h1(t+1)
    for (int i = 1; i < Nx - 1; i++) {
      for (int j = 1; j <= Ny - 1; j++) {
        h0[i * Ny + j] = h1[i * Ny + j];
        h1[i * Ny + j] = h2[i * Ny + j];
      }
    }
  }
}

int main() {
  // Print out specs of the main GPU
  hipDeviceProp_t deviceProp;
  CHECK(hipGetDeviceProperties(&deviceProp, 0));
  printf("GPU0:\t%s\t%d.%d:\n", deviceProp.name, deviceProp.major,
         deviceProp.minor);
  printf("\t%lu GB:\t total Global memory (gmem)\n",
         deviceProp.totalGlobalMem / 1024 / 1024 / 1000);
  printf("\t%lu MB:\t total Constant memory (cmem)\n",
         deviceProp.totalConstMem / 1024);
  printf("\t%lu MB:\t total Shared memory per block (smem)\n",
         deviceProp.sharedMemPerBlock / 1024);
  printf("\t%d:\t total threads per block\n", deviceProp.maxThreadsPerBlock);
  printf("\t%d:\t total registers per block\n", deviceProp.regsPerBlock);
  printf("\t%d:\t warp size\n", deviceProp.warpSize);
  printf("\t%d x %d x %d:\t max dims of block\n", deviceProp.maxThreadsDim[0],
         deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
  printf("\t%d x %d x %d:\t max dims of grid\n", deviceProp.maxGridSize[0],
         deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
  CHECK(hipSetDevice(0));

  double dx = 0.1;
  double dy = dx;
  double h = dx;          // 离散在一个均匀的网格 dx = dy = h
  int Nx = ceil(Lx / dx); // Nx : 40
  int Ny = ceil(Ly / dy); // Ny : 20
  double *h0, *h1, *h2;   // wave hight 波高
  h0 = (double *)calloc((Nx + 1) * (Ny + 1),
                        sizeof(double)); // t(n-1) 分配数据在栈空间
  h1 = (double *)calloc((Nx + 1) * (Ny + 1), sizeof(double)); // t(n)
  h2 = (double *)calloc((Nx + 1) * (Ny + 1), sizeof(double)); // t(n+1)

  // init
  // set the edge lines of the array to nmber 0 and
  // fill others values into slots
  for (int i = 0; i <= Nx; i++) {
    h1[i * (Ny + 1)] = 0;
    h1[i * (Ny + 1) + Ny] = 0;
  }
  for (int j = 0; j <= Ny; j++) {
    h1[j] = 0;
    h1[Nx * (Ny + 1) + j] = 0;
  }

  for (int i = 1; i <= Nx; i++) {
    for (int j = 1; j <= Ny; j++) {
      double x = h * i;
      double y = h * j;
      h1[i * (Ny+1) + j] = 0.1 * (4 * x - x * x) * (2 * y - y * y);
    }
  }

  // time step size should satify: dt < h / c
  double dt = h / (2 * c);
  

  // init h0
  for (int i = 1; i <= Nx - 1; i++) {
    for (int j = 1; j <= Ny - 1; j++) {
      h0[i * (Ny + 1) + j]=
          (c * c * dt * dt / (2 * h * h)) *
              (h1[(i + 1) * (Ny + 1) + j] + h1[(i - 1) * (Ny + 1) + j] +
               h1[i * (Ny + 1) + j + 1] + h1[i * (Ny + 1) + j - 1] -
               4 * h1[i * (Ny + 1) + j]);
    }
  }

  print_height(h1,Nx,Ny);

  // interatioral time step
  size_t nbytes =
      (Nx + 1) * (Ny + 1) * sizeof(double); // bytes to store nx * ny
  double *d_h0, *d_h1, *d_h2;
  CHECK(hipMalloc((void **)&d_h0, nbytes));
  CHECK(hipMalloc((void **)&d_h1, nbytes));
  CHECK(hipMalloc((void **)&d_h2, nbytes));
  CHECK(hipMemset(d_h2, 0, nbytes));
  CHECK(hipMemcpy(d_h0, h0, nbytes, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_h1, h1, nbytes, hipMemcpyHostToDevice));

  // Setup CUDA run
  dim3 block(128);
  dim3 grid((IT_NUM + block.x - 1)  / block.x);
  
  kernel_2dfd<<<grid, block>>>(d_h0, d_h1, d_h2, dt, h, Nx, Ny);
  CHECK(hipDeviceSynchronize());
  CHECK(hipMemcpy(d_h1, d_h1, nbytes, hipMemcpyDeviceToHost));
  

  // print_height(h1, Nx, Ny);

  // get the highest point
  double hmax = 0;
  int imax = 0, jmax = 0;
  for (int i = 1; i < Nx - 1; i++) {
    for (int j = 1; j <= Ny - 1; j++) {
      if (fabs(hmax) < fabs(h1[i * Nx + j])) {
        hmax = h1[i * Nx + j];
        imax = i;
        jmax = j;
      }
    }
  }
  cout << "hmax is " << hmax << " imax is " << imax << " jmax is " << jmax
       << endl;

  // get lines values from numerical results
  int Lx1_idx = static_cast<int>(0.25 * Lx / h);
  int Lx2_idx = static_cast<int>(0.50 * Lx / h);
  int Lx3_idx = static_cast<int>(0.75 * Lx / h);

  double *h_Lx1_numaric = (double *)calloc(Ny + 1, sizeof(double));
  double *h_Lx2_numaric = (double *)calloc(Ny + 1, sizeof(double));
  double *h_Lx3_numaric = (double *)calloc(Ny + 1, sizeof(double));

  for (int i = 0; i <= Ny; i++) {
    h_Lx1_numaric[i] = h1[Lx1_idx * Nx + i];
    h_Lx2_numaric[i] = h1[Lx2_idx * Nx + i];
    h_Lx3_numaric[i] = h1[Lx3_idx * Nx + i];
  }

  // for (size_t i = 0; i < Ny; i++) {
  //   cout << h_Lx1_numaric[i] << " " << h_Lx2_numaric[i] << " "
  //        << h_Lx3_numaric[i] << endl;
  // }

  double *h_Lx1_precise = (double *)calloc(Ny + 1, sizeof(double));
  double *h_Lx2_precise = (double *)calloc(Ny + 1, sizeof(double));
  double *h_Lx3_precise = (double *)calloc(Ny + 1, sizeof(double));

  double total_time = IT_NUM * dt;
  for (int i = 0; i <= Ny; i++) {
    h_Lx1_precise[i] = precise_solution((0.25 * Lx), (i * h), total_time);
    h_Lx2_precise[i] = precise_solution((0.50 * Lx), (i * h), total_time);
    h_Lx3_precise[i] = precise_solution((0.75 * Lx), (i * h), total_time);
  }
  // for (size_t i = 0; i < Ny; i++) {
  //   cout << h_Lx1_precise[i] << " " << h_Lx1_precise[i] << " "
  //        << h_Lx1_precise[i] << endl;
  // }

  CHECK(hipFree(d_h0));
  CHECK(hipFree(d_h1));
  CHECK(hipFree(d_h2));
  CHECK(hipDeviceReset());

  free(h0);
  free(h1);
  free(h2);
  free(h_Lx1_numaric);
  free(h_Lx2_numaric);
  free(h_Lx3_numaric);
  free(h_Lx1_precise);
  free(h_Lx2_precise);
  free(h_Lx3_precise);
  
  return 0;
  // * /
}

double **calloc2D(int x, int y) {

  double **u = (double **)calloc(x, sizeof(double));

  if (u == NULL) {
    printf(" memory cannot be allocated inside function memalloc2D");
    exit(-1);
  }

  for (int i = 0; i < x; ++i) {
    u[i] = (double *)calloc(y, sizeof(double));
    if (u[i] == NULL) {
      printf(" memory cannot be allocated");
      printf("i=%d\n", i);
      exit(-1);
    }
  }
  return u;
}

void free2D(double **u, int x, int y) {
  for (int i = 0; i < x; ++i) {
    free(u[i]);
  }
  free(u);
}
